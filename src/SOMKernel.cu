#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include "math/Random.h"
#include "math/Functions.h"
#include "Functors.h"
#include "SOMNetGPU.h"

#include <cfloat>
#include <cassert>
#include <cmath>
#include <algorithm>

#include <omp.h>

#include <thrust/extrema.h>
#include <thrust/distance.h>
#include <thrust/device_vector.h>

using namespace ANNGPGPU;


typedef float (*pDistanceFu) (float, float);
__device__ pDistanceFu pBubble 		= ANN::fcn_bubble_nhood; 
__device__ pDistanceFu pGaussian 	= ANN::fcn_gaussian_nhood; 
__device__ pDistanceFu pCutGauss 	= ANN::fcn_cutgaussian_nhood; 
__device__ pDistanceFu pMexican 	= ANN::fcn_mexican_nhood; 
__device__ pDistanceFu pEpanech 	= ANN::fcn_epanechicov_nhood;

bool SOMNetGPU::AssignDistanceFunction() {
	pDistanceFu hBubble; 
	pDistanceFu hGaussian; 
	pDistanceFu hCutGauss; 
	pDistanceFu hMexican; 
	pDistanceFu hEpanech;

	hipMemcpyFromSymbol(&hBubble, HIP_SYMBOL(pBubble), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hGaussian, HIP_SYMBOL(pGaussian), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hCutGauss, HIP_SYMBOL(pCutGauss), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hMexican, HIP_SYMBOL(pMexican), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hEpanech, HIP_SYMBOL(pEpanech), sizeof(pDistanceFu) );

	if (strcmp (GetDistFunction()->name, "gaussian") == 0) {
		GetDistFunction()->distance = hGaussian;
	} else if (strcmp (GetDistFunction()->name, "mexican") == 0) {
		GetDistFunction()->distance = hMexican;
	} else if (strcmp (GetDistFunction()->name, "bubble") == 0) {
		GetDistFunction()->distance = hBubble;
	} else if (strcmp (GetDistFunction()->name, "cutgaussian") == 0) {
		GetDistFunction()->distance = hCutGauss;
	} else if (strcmp (GetDistFunction()->name, "epanechicov") == 0) {
		GetDistFunction()->distance = hEpanech;
	} else {
		printf("No preimplemented function recognized. No assignment done.");
		return 0;
	}
	printf("Preimplemented function recognized. Assignment done.");
	return 1;
}

bool SOMNetGPU::DeassignDistanceFunction() {
	if (strcmp (GetDistFunction()->name, "gaussian") == 0) {
		GetDistFunction()->distance = ANN::fcn_gaussian_nhood; 
	} else if (strcmp (GetDistFunction()->name, "mexican") == 0) {
		GetDistFunction()->distance = ANN::fcn_mexican_nhood; 
	} else if (strcmp (GetDistFunction()->name, "bubble") == 0) {
		GetDistFunction()->distance = ANN::fcn_bubble_nhood;
	} else if (strcmp (GetDistFunction()->name, "cutgaussian") == 0) {
		GetDistFunction()->distance = ANN::fcn_cutgaussian_nhood;
	} else if (strcmp (GetDistFunction()->name, "epanechicov") == 0) {
		GetDistFunction()->distance = ANN::fcn_epanechicov_nhood;
	} else {
		printf("No preimplemented function recognized. No deassignment done.");
		return 0;
	}
	printf("Preimplemented function recognized. Deassignment done.");
	return 1;
}

// new reference implementation
ANNGPGPU::BMUExport hostGetMin(std::vector<ANNGPGPU::BMUExport> &vec) {
	assert(vec.size() > 0);
	if(vec.size() > 1) {
		std::sort(vec.begin(), vec.end() );
	}
	return *vec.begin();
}

// fast when maps are big
std::pair<float, unsigned int> devGetMin(const thrust::device_vector<float> &vec) {
	thrust::device_vector<float>::const_iterator d_min = thrust::min_element(vec.begin(), vec.end() );
	unsigned int iID = thrust::distance(vec.begin(), d_min);
	return std::pair<float, unsigned int>(*d_min, iID);
}

//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
/*
 * Layout of SOMEdgeF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
BMUExport
hostSOMFindBMNeuronID(std::vector<SOMExport*> &SExp,
		const float &fConscRate)
{
	BMUExport resBMU;
	std::vector<ANNGPGPU::BMUExport> vBMUExp(SExp.size() );

	assert(SExp.size() > 0);
	assert(vBMUExp.size() == SExp.size() );

	omp_set_num_threads(SExp.size() );  							// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 									// for(int iDevID = 0; iDevID < static_cast<int>(SExp.size() ); iDevID++) {
	{
		unsigned int iDevID 	= omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDevID) );
		
		unsigned int iWidth 	= SExp.at(iDevID)->f2dEdges.GetW();
		unsigned int iHeight 	= SExp.at(iDevID)->f2dEdges.GetH();

		assert(iWidth 	> 0);
		assert(iHeight 	> 0);

		thrust::device_vector<float> dvRes(iWidth, 0.f);

		for(int y = 0; y < static_cast<int>(iHeight); y++) {               
			thrust::transform(SExp.at(iDevID)->f2dEdges.GetRowBegin(y),
				SExp.at(iDevID)->f2dEdges.GetRowEnd(y),
				dvRes.begin(),
				dvRes.begin(),
				spowAmXpY_functor((*SExp.at(iDevID)->dvInput)[y]) );
		}

		if(fConscRate > 0.f) { 								// Implementation of conscience mechanism
			thrust::transform(dvRes.begin(),					// input
				dvRes.end(),							// input
				SExp.at(iDevID)->dvConscience->begin(),				// input
				dvRes.begin(),							// result
				sXmAmY_functor(1.f/(float)iWidth) );				// functor

			thrust::transform(dvRes.begin(),					// input
				dvRes.end(),							// input
				SExp.at(iDevID)->dvConscience->begin(),				// input
				SExp.at(iDevID)->dvConscience->begin(),				// result
				sAXmY_functor(fConscRate) );					// functor
		}

		std::pair<float, unsigned int> pCurBMUVal = devGetMin(dvRes);
		BMUExport BMU(pCurBMUVal.first, pCurBMUVal.second, iDevID);
		vBMUExp[iDevID] = BMU;
	}

	resBMU = hostGetMin(vBMUExp);
	checkCudaErrors(hipSetDevice(resBMU.iDeviceID) );
	resBMU.dvBMUPos = SExp.at(resBMU.iDeviceID)->f2dPositions.GetSubArrayY(resBMU.iBMUID);

	return resBMU;
}

/*
 * Layout of SOMPositionF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
template<typename BinaryFunction>
void hostSOMPropagateBW( std::vector<SOMExport*> &SExp,
		const BMUExport &BMU,
		const unsigned int &fCycle,
		const unsigned int &fCycles,
		BinaryFunction binaryDistFunc
		)
{
	omp_set_num_threads(SExp.size() );  							// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 									// for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
	{
		unsigned int iDevID 	= omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDevID) );
		
		unsigned int iWidth 	= SExp.at(iDevID)->f2dPositions.GetW();
		unsigned int iHeight 	= SExp.at(iDevID)->f2dPositions.GetH();

		thrust::device_vector<float> dvTmp (iWidth, 0.f); 				// temporary
		thrust::device_vector<float> dvLearningRate(iWidth, 0.f);
		thrust::device_vector<float> dvInfl(iWidth, 0.f);
		thrust::device_vector<float> dvDist(iWidth, 0.f);
		
		// 1. Calc distances for all neurons to BMNeuron: Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) )
		for(int y = 0; y < static_cast<int>(iHeight); y++) { 				// for each coordinate position of the neuron
			thrust::transform(
				SExp.at(iDevID)->f2dPositions.GetRowBegin(y),
				SExp.at(iDevID)->f2dPositions.GetRowEnd(y),
				dvDist.begin(),
				dvDist.begin(),
				spowAmXpY_functor(BMU.dvBMUPos[y]) );
		}

                thrust::transform(dvDist.begin(), dvDist.end(), dvDist.begin(), square_root());
		
		// 1 b calc learning rate
		thrust::device_vector<float> *dvLRate = SExp.at(iDevID)->dvLearningRate;
		thrust::transform( dvLRate->begin(),						// input
			dvLRate->end(), 							// input
			dvLearningRate.begin(), 						// result
			sm13lrate_decay_functor(fCycle, fCycles) );				// functor
			
		// 1 c Calc SigmaT
		thrust::device_vector<float> *dvSigma0 = SExp.at(iDevID)->dvSigma0;
		thrust::transform( dvSigma0->begin(),						// input
			dvSigma0->end(), 							// input
			dvTmp.begin(), 								// result
			sm13rad_decay_functor(fCycle, fCycles) );				// functor
		
		// 2. Calculate the influence for each neuron
		thrust::transform( dvTmp.begin(),						// input
			dvTmp.end(), 								// input
			dvDist.begin(), 							// input 2
			dvInfl.begin(), 							// result
			binaryDistFunc );							// functor

		// 2 b
		thrust::transform( dvInfl.begin(),						// input
			dvInfl.end(), 								// input
			dvLearningRate.begin(), 						// input 2
			dvInfl.begin(), 							// result
			thrust::multiplies<float>() );								// functor

		// 3. Only handle neurons in radius:
		// 3a. Make stencil
		thrust::transform( dvDist.begin(), 						// input
			dvDist.end(),								// input
			dvTmp.begin(),								// input 2
			dvTmp.begin(), 								// result
			thrust::less<float>() 							// functor
		);
		// 3b. Use stencil to modify only neurons inside the radius
		iWidth 	= SExp.at(iDevID)->f2dEdges.GetW();
		iHeight = SExp.at(iDevID)->f2dEdges.GetH();
		for(int y = 0; y < static_cast<int>(iHeight); y++) {				// for each edge of the neuron
			thrust::transform_if( SExp.at(iDevID)->f2dEdges.GetRowBegin(y),		// input 1
				SExp.at(iDevID)->f2dEdges.GetRowEnd(y), 			// input 1
				dvInfl.begin(),							// input 2
				dvTmp.begin(),							// stencil
				SExp.at(iDevID)->f2dEdges.GetRowBegin(y), 			// result
				hebbian_functor((*SExp.at(iDevID)->dvInput)[y]), // functor
				thrust::identity<int>() ); 					// predicate
		}
	}
}

void hostSOMTrainHelper( std::vector<SOMExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles, 
		const float &fConscRate,
		const ANN::DistFunction &DistFunc, 
		const unsigned int &iPatternID,
		const unsigned int &iCycle) 
{
	assert(iPatternID < InputSet.GetNrElements() );

	// Set Input
	std::vector<float> vCurInput = InputSet.GetInput(iPatternID);
	for(int iDevID = 0; iDevID < static_cast<int>(SExp.size() ); iDevID++) {
		checkCudaErrors(hipSetDevice(iDevID) );

		thrust::device_vector<float> *p_dvInputVector = new thrust::device_vector<float>(vCurInput.size() );
		thrust::copy(vCurInput.begin(), vCurInput.end(), p_dvInputVector->begin() );
		SExp[iDevID]->dvInput = p_dvInputVector;
	}

	// Find BMNeuron 
	BMUExport BMUExp = hostSOMFindBMNeuronID(SExp, fConscRate);

	// Propagate BW SM 2.0
	hostSOMPropagateBW( SExp,
		BMUExp,									// const
		iCycle,
		iCycles,
		sm20distance_functor(DistFunc.distance)); 				// const
}

void hostSOMTraining( std::vector<SOMExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles, 
		const float &fConscRate,
		const ANN::DistFunction &DistFunc,
		const ANN::TrainingMode &eMode )
{
	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	int iProgCount 		= 1;

	for(int iCycle = 0; iCycle < static_cast<int>(iCycles); iCycle++) {
		if(iCycles >= 10) {
			if(((iCycle+1) / (iCycles/10)) == iProgCount && (iCycle+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<iCycle+1<<std::endl;
				iProgCount++;
			}
		} 
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(iCycle+1.f)/(float)iCycles*100.f<<"%/Step="<<iCycle+1<<std::endl;
		}

		if(eMode == ANN::ANRandomMode) {
			unsigned int iRandID = ANN::RandInt(iMin, iMax);
			hostSOMTrainHelper(SExp, InputSet, iCycles, fConscRate, DistFunc, iRandID, iCycle);
		}
		// The input vectors are presented to the network in serial order
		else if(eMode == ANN::ANSerialMode) {
			for(unsigned int j = 0; j < InputSet.GetNrElements(); j++) {
				hostSOMTrainHelper(SExp, InputSet, iCycles, fConscRate, DistFunc, j, iCycle);
			}
		}
	}
}

#endif
