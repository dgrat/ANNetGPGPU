#include "hip/hip_runtime.h"
#include "SetFcn.h"


typedef float (*pDistanceFu) (float, float);
typedef float (*pDecayFu) (float, float, float);

// Custom Guassian that falls off to 0 (default falls off to 0.6)
__device__ static float distanceFunction(float dist, float sigmaT) {
    //return -(dist/sigmaT)+1;
	return exp(-pow(dist, 2.f)/(0.25f*pow(sigmaT, 2.f)));
    //return 0.5;
    /*float value = -(dist/sigmaT)+1;
    if (value < 0)
        value = 0;
    else if (value > 1)
        value = 1;
    return value;*/
}

__device__ static float distanceDecay (float sigma0, float T, float lambda) {
	return std::floor(sigma0*exp(-T/lambda) + 0.5f);
}

__device__ pDistanceFu pOwn = distanceFunction; 
__device__ pDecayFu pOwn2 = distanceDecay;

void SetFcn(ANN::DistFunction *fcn) {
	pDistanceFu hOwn;
        pDecayFu hOwn2;
	hipMemcpyFromSymbol(&hOwn, HIP_SYMBOL(pOwn), sizeof(pDistanceFu) );
        hipMemcpyFromSymbol(&hOwn2, HIP_SYMBOL(pOwn2), sizeof(pDecayFu) );
	fcn->distance = hOwn;
        fcn->rad_decay = hOwn2;
}
